#include "util.h"
#include "cuda_util.h"
#include "cub.cuh"
#include "cub_sort.cuh"
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>


void cubRadixSortKey(uint64_t *d_keys_in, uint64_t *d_keys_out, 
                     uint32_t numOfEntry)
{
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                   d_keys_in, d_keys_out, numOfEntry);
    // allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // run sorting operation
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                   d_keys_in, d_keys_out, numOfEntry);
}

void cubRadixSortInterval(d_Gaps &d_intervals_in, d_Gaps &d_intervals_out, 
                          uint32_t numOfInterval)
{
    // with junctions
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_intervals_in.start_, d_intervals_out.start_,
                                    d_intervals_in.end_, d_intervals_out.end_,
                                    numOfInterval);
    // allocate temporary storage
    CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_intervals_in.start_, d_intervals_out.start_,
                                    d_intervals_in.end_, d_intervals_out.end_,
                                    numOfInterval);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void cubRadixSortJunction(d_Junctions &d_junctions_in, d_Junctions &d_junctions_out,
                             h_Junctions &h_junctions, uint32_t numOfJunction)
{
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                   d_junctions_in.end_, d_junctions_out.end_,
                                   d_junctions_in.start_, d_junctions_out.start_,
                                   numOfJunction);
    // allocate temporary storage
    CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                   d_junctions_in.end_, d_junctions_out.end_,
                                   d_junctions_in.start_, d_junctions_out.start_,
                                   numOfJunction);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    d_temp_storage = nullptr;
    temp_storage_bytes = 0;
    // determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_junctions_out.start_, d_junctions_in.start_,
                                    d_junctions_out.end_, d_junctions_in.end_,
                                    numOfJunction);
    // allocate temporary storage
    CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_junctions_out.start_, d_junctions_in.start_,
                                    d_junctions_out.end_, d_junctions_in.end_,
                                    numOfJunction);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    CUDA_SAFE_CALL(hipMemcpy(d_junctions_out.start_, d_junctions_in.start_,
                              sizeof(uint64_t) * numOfJunction,
                              hipMemcpyDeviceToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_junctions_out.end_, d_junctions_in.end_,
                              sizeof(uint64_t) * numOfJunction,
                              hipMemcpyDeviceToDevice));
}


d_Junctions thrustSegmentedScanJunction(d_Junctions &d_junctions_in, uint32_t &numOfJunction)
{
    // segmented prefix sum
    thrust::device_vector<uint32_t> d_counts_start(numOfJunction);
    thrust::device_vector<uint32_t> d_counts_end(numOfJunction);
    thrust::fill(thrust::device, d_counts_start.begin(), d_counts_start.end(), 1);
    thrust::fill(thrust::device, d_counts_end.begin(), d_counts_end.end(), 1);
    thrust::inclusive_scan_by_key(thrust::device, d_junctions_in.start_,
                                  d_junctions_in.start_ + numOfJunction,
                                  d_counts_start.begin(), d_counts_start.begin());
    thrust::inclusive_scan_by_key(thrust::device, d_junctions_in.end_,
                                  d_junctions_in.end_ + numOfJunction,
                                  d_counts_end.begin(), d_counts_end.begin());
    
    thrust::device_vector<uint32_t> d_counts(numOfJunction);
    thrust::transform(thrust::device, d_counts_start.begin(), d_counts_start.end(),
                      d_counts_end.begin(), d_counts.begin(), min_element<uint32_t>());

    // set flags
    thrust::device_vector<uint32_t> d_flags(numOfJunction);
    thrust::replace_copy_if(thrust::device, d_counts.begin(), d_counts.end(),
                            d_flags.begin(), is_greater_than_one<uint32_t>(), 0);

    // compute offsets
    thrust::device_vector<uint32_t> d_indices(numOfJunction);
    thrust::inclusive_scan(thrust::device, d_flags.begin(),
                           d_flags.end(), d_indices.begin());

    // calculate new numOfJunction
    uint32_t new_numOfJunction;
    CUDA_SAFE_CALL(hipMemcpy(&new_numOfJunction,
                              thrust::raw_pointer_cast(d_indices.data()) + numOfJunction - 1,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

    d_Junctions d_junctions_out;
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_junctions_out.start_, sizeof(uint64_t) * new_numOfJunction));
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_junctions_out.end_, sizeof(uint64_t) * new_numOfJunction));
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_junctions_out.count, sizeof(uint32_t) * new_numOfJunction));

    // compute number of junction block
    unsigned nJunctionBlock = ceilDiv(numOfJunction, blockSize);

    scatter_if<uint64_t><<<nJunctionBlock, blockSize>>>(
                        thrust::raw_pointer_cast(d_indices.data()),
                        d_junctions_in.start_, d_junctions_out.start_,
                        thrust::raw_pointer_cast(d_flags.data()),
                         numOfJunction);
    scatter_if<uint64_t><<<nJunctionBlock, blockSize>>>(
                        thrust::raw_pointer_cast(d_indices.data()),
                        d_junctions_in.end_, d_junctions_out.end_,
                        thrust::raw_pointer_cast(d_flags.data()),
                        numOfJunction);
    scatter_if<uint32_t><<<nJunctionBlock, blockSize>>>(
                        thrust::raw_pointer_cast(d_indices.data()),
                        thrust::raw_pointer_cast(d_counts.data()),
                        d_junctions_out.count,
                        thrust::raw_pointer_cast(d_flags.data()),
                        numOfJunction);
    // update numOfJunction
    numOfJunction = new_numOfJunction;

    return d_junctions_out;
}

// sort bins by using cub library
void cubRadixSortBin(d_Bins &d_bins_in, d_Bins &d_bins_out, 
                      h_Bins &h_bins, uint32_t numOfBin)
{
    // indices on cpu
    auto *indices = new uint32_t[numOfBin];
    std::iota(indices, indices+numOfBin, 0);

    // indices on gpu
    uint32_t *d_indices_in;
    uint32_t *d_indices_out;
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_indices_in, sizeof(uint32_t) * numOfBin));
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_indices_out, sizeof(uint32_t) * numOfBin));
    CUDA_SAFE_CALL(
        hipMemcpy(d_indices_in, indices, sizeof(uint32_t) * numOfBin,
                   hipMemcpyHostToDevice));

    // with junctions
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    //! determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_bins_in.start_, d_bins_out.start_,
                                    d_indices_in, d_indices_out, numOfBin);
    // allocate temporary storage
    CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_bins_in.start_, d_bins_out.start_,
                                    d_indices_in, d_indices_out, numOfBin);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // compute number of thread block for bins
    unsigned nBinBlock = ceilDiv(numOfBin, blockSize);

    gather<uint64_t><<<nBinBlock, blockSize>>>(d_indices_out, d_bins_in.end_,
                                               d_bins_out.end_, numOfBin);
    gather<uint8_t><<<nBinBlock, blockSize>>>(d_indices_out, d_bins_in.strand,
                                              d_bins_out.strand, numOfBin);
    gather<bin_core_t><<<nBinBlock, blockSize>>>(d_indices_out, d_bins_in.core,
                                                 d_bins_out.core, numOfBin);
    CUDA_SAFE_CALL(hipMemcpy(h_bins.start_.data(), d_bins_out.start_, 
                        numOfBin * sizeof(uint64_t), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(h_bins.end_.data(), d_bins_out.end_, 
                        numOfBin * sizeof(uint64_t), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(h_bins.strand.data(), d_bins_out.strand,
                        numOfBin * sizeof(uint8_t), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(h_bins.core.data(), d_bins_out.core,
                        numOfBin * sizeof(bin_core_t), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // free used memory
    delete []indices;
    CUDA_SAFE_CALL(hipFree(d_indices_in));
    CUDA_SAFE_CALL(hipFree(d_indices_out));
}

// sort ases by using cub library
void cubRadixSortASE(d_ASEs &d_ases_in, d_ASEs &d_ases_out,
                     uint32_t numOfASE)
{
    // indices on cpu
    auto *indices = new uint32_t[numOfASE];
    std::iota(indices, indices+numOfASE, 0);

    // indices on gpu
    uint32_t *d_indices_in;
    uint32_t *d_indices_out;
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_indices_in, sizeof(uint32_t) * numOfASE));
    CUDA_SAFE_CALL(
        hipMalloc((void **)&d_indices_out, sizeof(uint32_t) * numOfASE));
    CUDA_SAFE_CALL(
        hipMemcpy(d_indices_in, indices, sizeof(uint32_t) * numOfASE,
                   hipMemcpyHostToDevice));

    // with junctions
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    //! determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_ases_in.start_, d_ases_out.start_,
                                    d_indices_in, d_indices_out, numOfASE);
    // allocate temporary storage
    CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    d_ases_in.start_, d_ases_out.start_,
                                    d_indices_in, d_indices_out, numOfASE);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // compute number of thread block for ases
    unsigned nASEBlock = ceilDiv(numOfASE, blockSize);

    gather<uint64_t><<<nASEBlock, blockSize>>>(d_indices_out, d_ases_in.end_,
                                               d_ases_out.end_, numOfASE);
    gather<uint8_t><<<nASEBlock, blockSize>>>(d_indices_out, d_ases_in.strand,
                                              d_ases_out.strand, numOfASE);
    gather<ase_core_t><<<nASEBlock, blockSize>>>(d_indices_out, d_ases_in.core,
                                                 d_ases_out.core, numOfASE);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // free used memory
    delete []indices;
    CUDA_SAFE_CALL(hipFree(d_indices_in));
    CUDA_SAFE_CALL(hipFree(d_indices_out));
}

// cub reduce sum
void cubReduceSum(float *d_in, float *d_out, uint32_t num_items)
{
    // determine temporary device storage requirements
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    int num_items_ = int(num_items);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, 
                           d_in, d_out, num_items_);
    // allocate temporary storage
    CUDA_SAFE_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // run sum-reduction
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, 
                           d_in, d_out, num_items_);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}
